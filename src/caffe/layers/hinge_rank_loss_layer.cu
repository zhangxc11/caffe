#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {

template <typename Dtype>
__global__ void kernel_get_lab_val(const int num, const int dim,
    const Dtype* label, const Dtype* margin, Dtype* data, Dtype* val) {
  CUDA_KERNEL_LOOP(index, num) {
    val[index] = data[index * dim + static_cast<int>(label[index])];
    // the loss for labeled data is zero
    data[index * dim + static_cast<int>(label[index])] -= *margin;
  }
}

template <typename Dtype>
__global__ void kernel_hinge_max(const int num, const int dim,
    const Dtype* labelval, const Dtype* margin, Dtype* data) {
  CUDA_KERNEL_LOOP(index, num * dim){
    int n = index / dim;
    data[index] = max(Dtype(0), *margin - labelval[n] + data[index]);
  }
}

template <typename Dtype>
__global__ void kernel_set_diff(const int num, const int dim,
    const Dtype* label, const Dtype* val, Dtype* diff) {
  CUDA_KERNEL_LOOP(index, num) {
    diff[index * dim + static_cast<int>(label[index])] = val[index];
  }
}

template <typename Dtype>
Dtype HingeRankLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* labval_data = labval_.mutable_gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const Dtype* margin_data = margin_.gpu_data();
  int num = bottom[0]->num();
  int count = bottom[0]->count();
  int dim = count / num;
  caffe_gpu_copy(count, bottom_data, bottom_diff);
  kernel_get_lab_val<Dtype><<<CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS>>>(
      num, dim, label, margin_data, bottom_diff, labval_data);
  kernel_hinge_max<Dtype><<<CAFFE_GET_BLOCKS(num * dim),
    CAFFE_CUDA_NUM_THREADS>>>(num, dim, labval_data, margin_data, bottom_diff);
  Dtype sum;
  caffe_gpu_asum(count, bottom_diff, &sum);
  return  sum / num;
}

template <typename Dtype>
void HingeRankLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  const Dtype* label = (*bottom)[1]->gpu_data();
  Dtype* labval_data = labval_.mutable_gpu_data();
  int num = (*bottom)[0]->num();
  int count = (*bottom)[0]->count();
  int dim = count / num;

  caffe_gpu_sign(count, bottom_diff, bottom_diff);
  caffe_gpu_gemv<Dtype>(CblasNoTrans, num, dim, -1., bottom_diff,
      sum_multiplier_.gpu_data(), 0., labval_data);
  kernel_set_diff<Dtype><<<CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS>>>(
      num, dim, label, labval_data, bottom_diff);
  caffe_gpu_scal(count, Dtype(1. / num), bottom_diff);
}

INSTANTIATE_CLASS(HingeRankLossLayer);


}  // namespace caffe
