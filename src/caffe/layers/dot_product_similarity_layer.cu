#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {

template <typename Dtype>
__global__ void kernel_get_max(const int num, const int dim,
    const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num) {
    Dtype maxval = -FLT_MAX;
    for (int i = 0; i < dim; ++i) {
      maxval = max(data[index * dim + i], maxval);
    }
    out[index] = maxval;
  }
}

template <typename Dtype>
__global__ void kernel_dotp_div(const int num, const int dim,
    const Dtype* scale, Dtype* data) {
  CUDA_KERNEL_LOOP(index, num * dim) {
    int n = index / dim;
    data[index] /= scale[n];
  }
}


template <typename Dtype>
Dtype DotProductSimilarityLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  Dtype* scale_data = scale_.mutable_gpu_data();
  const Dtype* weight = this->simvec_.gpu_data();
  // kernel_get_max<Dtype><<<CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS>>>(
  //    num, dim, bottom_data, scale_data);
  // do inner product
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
      bottom_data, weight, (Dtype)0., top_data);
  // uniform vectors
  // sum
  caffe_gpu_gemv<Dtype>(CblasNoTrans, M_, K_, 1., bottom_data,
      sum_multiplier_.gpu_data(), 0., scale_data);
  // Do division
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_dotp_div<Dtype><<<CAFFE_GET_BLOCKS(M_ * N_),
                              CAFFE_CUDA_NUM_THREADS>>>(
      M_, N_, scale_data, top_data);
  return Dtype(0);
}

template <typename Dtype>
void DotProductSimilarityLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down) return;
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* scale_data = scale_.gpu_data();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
      top_diff, this->simvec_.gpu_data(), (Dtype)0., bottom_diff);
  // Compute inner1d(top_diff, top_data) and subtract them from the bottom diff
  // cuda dot returns the result to cpu, so we temporarily change the pointer
  // mode
  CUBLAS_CHECK(hipblasSetPointerMode(Caffe::cublas_handle(),
      HIPBLAS_POINTER_MODE_DEVICE));
  Dtype* sub_data = sub_.mutable_gpu_data();
  for (int i = 0; i < M_; ++i) {
    caffe_gpu_dot<Dtype>(N_, top_diff + i * N_,
        top_data + i * N_, sub_data + i);
  }
  CUBLAS_CHECK(hipblasSetPointerMode(Caffe::cublas_handle(),
      HIPBLAS_POINTER_MODE_HOST));
  // subtraction
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, 1, -1.,
      sub_.gpu_data(), sum_multiplier_.gpu_data(), 1., bottom_diff);
  // Do division
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_dotp_div<Dtype><<<CAFFE_GET_BLOCKS(M_ * K_),
                              CAFFE_CUDA_NUM_THREADS>>>(
      M_, K_, scale_data, bottom_diff);
}

INSTANTIATE_CLASS(DotProductSimilarityLayer);


}  // namespace caffe
